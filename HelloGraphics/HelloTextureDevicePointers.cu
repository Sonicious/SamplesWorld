#include "hip/hip_runtime.h"
// other Headers
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "glad/glad.h"
#include <GLFW/glfw3.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

// GL Defines for stuff
constexpr int GL_VERTEX_POSITION_ATTRIBUTE = 0;
constexpr int GL_VERTEX_TEXTURE_COORD_ATTRIBUTE = 1;

void checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        printf("[ERROR] %s\n", hipGetErrorName(result));
        printf("[ERROR] %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void checkGL()
{
  GLenum error = glGetError();
  if(error != GL_NO_ERROR)
  {
    printf("[ERROR] %d\n", error);
    exit(EXIT_FAILURE);
  }
}

void glfwErrorCallback(int error, const char* description)
{
  printf("Error: %s\n", description);
}

typedef struct {
  GLfloat position[3];
  GLfloat textureCoords[2];
} VertexData;

typedef struct {
  unsigned char r;
  unsigned char g;
  unsigned char b;
  unsigned char a;
} pixelRGBA;

///////////////////////////////////////////////////////////////////////////////
// CUDA Kernel for image:

__global__ void myTextureKernel(pixelRGBA *renderImageData, size_t width, size_t height, size_t pitch)
{
  for (int idy = blockIdx.y * blockDim.y + threadIdx.y;
         idy < height;
         idy += blockDim.y * gridDim.y)
      {
        for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
               idx < width;
               idx += blockDim.x * gridDim.x) 
            {
                // according to CUDA documentation (see hipMallocPitch())
                pixelRGBA *myPixel = (pixelRGBA*) ((char*)renderImageData + idy*pitch) + idx;
                myPixel->r = 128;
                myPixel->g = 120;
                myPixel->b = 80;
                myPixel->a = 255;
            }
      }
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void FramebufferSizeCallback(GLFWwindow* window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and 
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}

// Vertex Shader Source:
// input comes to "in vec3 aPos"
// output goes to "gl_position
const GLchar *vertexShaderSource =
  "#version 450 core\n"
  "layout (location = 0) in vec3 aPos;\n"
  "layout (location = 1) in vec2 aTexCoord;\n"
  "out vec2 TexCoord;\n"
  "void main()\n"
  "{\n"
  "   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
  "   TexCoord = aTexCoord;\n"
  "}\0";

// Fragment Shader Source:
// out declares output
// output is always FragColor
const GLchar *fragmentShaderSource =
  "#version 450 core\n"
  "out vec4 FragColor;\n"
  "in vec2 TexCoord;\n"
  "uniform sampler2D ourTexture;\n"
  "void main()\n"
  "{\n"
  "   FragColor = texture(ourTexture, TexCoord);\n"
  "}\n\0";

int main(int argc, char *argv[])
{
  // read TextureSize
  int textureWidth = 128;
  int textureHeight = 128;
  if (argc >= 3)
  {
    textureWidth = atoi(argv[1]);
    textureHeight = atoi(argv[2]);
  }

  // OpenGL Status Variables:
  GLint  success;
  char infoLog[512];

///////////////////////////////////////////////////////////////////////////////
// Initialize everything for GLFW

  // Initialize the library
  if (!glfwInit())
  {
    return EXIT_FAILURE;
  }
  // Set Error Callback
  glfwSetErrorCallback(glfwErrorCallback);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4); // We want OpenGL 4.5
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  // Create a windowed mode window and its OpenGL context
  GLFWwindow* window = glfwCreateWindow(1000, 1000, "Hello Cuda GLFW Interop", NULL, NULL);
  if (!window)
  {
    printf("Failed to create GLFW window!");
    glfwTerminate();
    return EXIT_FAILURE;
  }
  // Make the window's context current
  glfwMakeContextCurrent(window);
  // load pointers to OpenGL functions at runtime
  if (!gladLoadGLLoader((GLADloadproc) glfwGetProcAddress))
  {
      printf("Failed to initialize OpenGL context");
      return EXIT_FAILURE;
  }
  // Manage Callbacks:
  glfwSetFramebufferSizeCallback(window, FramebufferSizeCallback);
    // disable Vsync
  glfwSwapInterval(0);

///////////////////////////////////////////////////////////////////////////////
// Create a state driven VAO
  GLuint VAO;
  glGenVertexArrays(1, &VAO);
  glBindVertexArray(VAO); // Bind Vertex Array First

///////////////////////////////////////////////////////////////////////////////
// Create Shader Program:
  
  // Create vertex shader and load and compile source
  GLuint vertexShader;
  vertexShader = glCreateShader(GL_VERTEX_SHADER);
  glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
  glCompileShader(vertexShader);
  // check for errors in compilation process
  glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
  if(!success)
  {
    glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
    printf("Vertex Shader Compilation Failed:\n%s\n",infoLog);
    return EXIT_FAILURE;
  }
  // Create fragment shader, load and compile
  GLuint fragmentShader;
  fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
  glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
  glCompileShader(fragmentShader);
  // check for errors in compilation process
  glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
  if(!success)
  {
    glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
    printf("Fragment Shader Compilation Failed:\n%s\n",infoLog);
    return EXIT_FAILURE;
  }
  // Link Shaders together to a program
  GLuint shaderProgram;
  shaderProgram = glCreateProgram();
  glAttachShader(shaderProgram, vertexShader);
  glAttachShader(shaderProgram, fragmentShader);
  glLinkProgram(shaderProgram);
  glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
  if(!success)
  {
      glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
      printf("Program Linking Failed:\n%s\n",infoLog);
      return EXIT_FAILURE;
  }

  // delete Shaders (not needed anymore, because completely compiled and so on)
  glDeleteShader(vertexShader);
  glDeleteShader(fragmentShader);

///////////////////////////////////////////////////////////////////////////////
// Setup Vertex Data, Buffers and configure Vertex Attributes:

  VertexData vertices[] = {
    {{-1.0f, -1.0f, 0.0f},{0.0f, 0.0f}},
    {{ 1.0f, -1.0f, 0.0f},{1.0f, 0.0f}},
    {{-1.0f,  1.0f, 0.0f},{0.0f, 1.0f}},
    {{ 1.0f,  1.0f, 0.0f},{1.0f, 1.0f}}
  };
  // generate buffer and Array for vertices and bind and fill it
  GLuint VBO;// Vertex Buffer Object, Vertex Array Object
  glGenBuffers(1, &VBO);
  glBindBuffer(GL_ARRAY_BUFFER, VBO);
  // Copy Vertices Data
  glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
  // Explain Data via VertexAttributePointers to the shader
  // By Default, it's disabled
  // Enable the Vertex Attribute
  glEnableVertexAttribArray(GL_VERTEX_POSITION_ATTRIBUTE);
  glVertexAttribPointer(GL_VERTEX_POSITION_ATTRIBUTE, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (GLvoid*)0);
  // Same for Texture: Pay Attention of Stride and begin
  glEnableVertexAttribArray(GL_VERTEX_TEXTURE_COORD_ATTRIBUTE);
  glVertexAttribPointer(GL_VERTEX_TEXTURE_COORD_ATTRIBUTE, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (GLvoid*)(3 * sizeof(float)));

  // possible unbinding:
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindVertexArray(0);

  // uncomment this call to draw in wireframe polygons.
  //glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

///////////////////////////////////////////////////////////////////////////////
// CUDA Texture Interaction

  GLuint interopTexture;
  pixelRGBA *deviceTextureGraphic;
  size_t deviceTextureGraphicPitch;
  hipGraphicsResource_t textureGraphicResource = 0;
  hipArray *textureCudaArray;


  // calculate Data size and MemAlloc Cuda Buffer
  checkCuda( hipMallocPitch(&deviceTextureGraphic, &deviceTextureGraphicPitch, textureWidth * sizeof(pixelRGBA), textureHeight) );
  
  // Here the Calculations for the interop-Data
  glGenTextures(1, &interopTexture);
  glBindTexture(GL_TEXTURE_2D, interopTexture);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);	
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  // Just allocate, but no copy to it:
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, textureWidth, textureHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
  // Register OpenGL texture to a hipGraphicsResource for CUDA
  checkCuda( hipGraphicsGLRegisterImage( &textureGraphicResource, interopTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

///////////////////////////////////////////////////////////////////////////////
// Render Loop
  double lastTime = glfwGetTime();
  int nbFrames = 0;
  char windowTitle[100];

  while (!glfwWindowShouldClose(window))
  {
    // Measure speed
    double currentTime = glfwGetTime();
    nbFrames++;
    if ( currentTime - lastTime >= 1.0 ){ // If last prinf() was more than 1 sec ago
        // printf and reset timer
        sprintf(windowTitle, "Hello Cuda GLFW Interop | %f ms/frame", 1000.0/double(nbFrames));
        glfwSetWindowTitle(window, windowTitle);
        nbFrames = 0;
        lastTime += 1.0;
    }

    // set bg color here via Clearing
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    // run CUDA
    myTextureKernel<<<16, 32>>>(deviceTextureGraphic, textureWidth, textureHeight, deviceTextureGraphicPitch);
    // Map 1 graphics resource for access by CUDA in stream 0
    checkCuda( hipGraphicsMapResources(1, &textureGraphicResource, 0) );
    // get the corresponding CudaArray of Resource at array position 0 and mipmap level 0
    checkCuda( hipGraphicsSubResourceGetMappedArray(&textureCudaArray, textureGraphicResource, 0, 0) );
    // copy Data to CudaArray from deviceRenderBuffer, wOffset=0, hOffset=0
    checkCuda( hipMemcpy2DToArray(textureCudaArray, 0, 0, deviceTextureGraphic, deviceTextureGraphicPitch, textureWidth * sizeof(pixelRGBA), textureHeight, hipMemcpyDeviceToDevice));
    // Unmap 1 resource from Stream 0
    checkCuda( hipGraphicsUnmapResources(1, &textureGraphicResource, 0) );

    // Draw
    // Use the program for the pipeline (keep it to save state to VAO)
    glUseProgram(shaderProgram);
    glBindVertexArray(VAO); checkGL(); // Program is bound to VAO
    glDrawArrays(GL_TRIANGLE_STRIP, 0, 4); // All about the loaded VAO
    glBindVertexArray(0); // To unbind Vertex Array
    glUseProgram(0);

    // Swap front and back buffers
    glfwSwapBuffers(window);

    // Check for Inputs:
    processInput(window);
    // Poll for and process events
    glfwPollEvents();
  }

  // Cleanup
  // OpenGL is reference counted and terminated by GLFW
  checkCuda( hipGraphicsUnregisterResource(textureGraphicResource) );
  checkCuda( hipFree(deviceTextureGraphic) );
  checkCuda( hipDeviceReset() );
  glfwDestroyWindow(window);
  glfwTerminate();
  return EXIT_SUCCESS;
}
