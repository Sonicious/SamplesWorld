#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#define GL_GLEXT_PROTOTYPES
#include <GLFW/glfw3.h>
#include <GL/gl.h>

// CUDA headers
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// GL Defines for stuff
#define GL_VERTEX_POSITION_ATTRIBUTE 0
#define GL_VERTEX_TEXTURE_COORD_ATTRIBUTE 1

inline void checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        printf("[ERROR] %s\n", hipGetErrorName(result));
        printf("[ERROR] %s\n", hipGetErrorString(result));
    }
}

typedef struct {
  unsigned char r;
  unsigned char g;
  unsigned char b;
  unsigned char a;
} pixelRGBA;

///////////////////////////////////////////////////////////////////////////////
// CUDA Kernel for image:

__global__ void myTextureKernel(unsigned char *renderImageData, size_t width, size_t height, size_t pitch)
{
  for (int idy = blockIdx.y * blockDim.y + threadIdx.y;
         idy < height;
         idy += blockDim.y * gridDim.y)
      {
        for (int idx = blockIdx.x * blockDim.x + threadIdx.x;
               idx < width;
               idx += blockDim.x * gridDim.x) 
            {
                renderImageData[idx*4+idy*pitch] = 255;
                //renderImageData[idx+idy*pitch].g = 255;
                //renderImageData[idx+idy*pitch].b = 255;
                //renderImageData[idx+idy*pitch].a = 255;
                printf("id: x:%i,y:%i\t",idx,idy);
            }
      }
  printf("\n");
}

///////////////////////////////////////////////////////////////////////////////
// IO-Callbacks:

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and 
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}

// Vertex Shader Source:
// input comes to "in vec3 aPos"
// output goes to "gl_position
const GLchar *vertexShaderSource = "#version 330 core\n"
  "layout (location = 0) in vec3 aPos;\n"
  "layout (location = 1) in vec2 aTexCoord;\n"
  "out vec2 TexCoord;\n"
  "void main()\n"
  "{\n"
  "   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
  "   TexCoord = aTexCoord;\n"
  "}\0";

// Fragment Shader Source:
// out declares output
// output is always FragColor
const GLchar *fragmentShaderSource = "#version 330 core\n"
  "out vec4 FragColor;\n"
  "in vec2 TexCoord;\n"
  "uniform sampler2D ourTexture;\n"
  "void main()\n"
  "{\n"
  "   FragColor = texture(ourTexture, TexCoord);\n"
  "}\n\0";

void error_callback(int error, const char* description)
{
  printf("Error: %s\n", description);
}

int main(void)
{
  // OpenGL Status Variables:
  GLint  success;
  char infoLog[512];

///////////////////////////////////////////////////////////////////////////////
// Initialize everything for GLFW

  // Initialize the library
  if (!glfwInit())
  {
    return EXIT_FAILURE;
  }
  // Set Error Callback
  glfwSetErrorCallback(error_callback);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3); // We want OpenGL 3.3
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  // Create a windowed mode window and its OpenGL context
  GLFWwindow* window = glfwCreateWindow(640, 480, "Hello Cuda GLFW Interop", NULL, NULL);
  if (!window)
  {
    printf("Failed to create GLFW window!");
    glfwTerminate();
    return EXIT_FAILURE;
  }
  // Make the window's context current
  glfwMakeContextCurrent(window);
  // Manage Callbacks:
  glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

///////////////////////////////////////////////////////////////////////////////
// Create a state driven VAO
  GLuint VAO;
  glGenVertexArrays(1, &VAO);
  glBindVertexArray(VAO); // Bind Vertex Array First

///////////////////////////////////////////////////////////////////////////////
// Create Shader Program:
  
  // Create vertex shader and load and compile source
  GLuint vertexShader;
  vertexShader = glCreateShader(GL_VERTEX_SHADER);
  glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
  glCompileShader(vertexShader);
  // check for errors in compilation process
  glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
  if(!success)
  {
    glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
    printf("Vertex Shader Compilation Failed:\n%s\n",infoLog);
    return EXIT_FAILURE;
  }
  // Create fragment shader, load and compile
  GLuint fragmentShader;
  fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
  glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
  glCompileShader(fragmentShader);
  // check for errors in compilation process
  glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
  if(!success)
  {
    glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
    printf("Fragment Shader Compilation Failed:\n%s\n",infoLog);
    return EXIT_FAILURE;
  }
  // Link Shaders together to a program
  GLuint shaderProgram;
  shaderProgram = glCreateProgram();
  glAttachShader(shaderProgram, vertexShader);
  glAttachShader(shaderProgram, fragmentShader);
  glLinkProgram(shaderProgram);
  glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
  if(!success)
  {
      glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
      printf("Program Linking Failed:\n%s\n",infoLog);
      return EXIT_FAILURE;
  }

  // delete Shaders (not needed anymore, because completely compiled and so on)
  glDeleteShader(vertexShader);
  glDeleteShader(fragmentShader);

///////////////////////////////////////////////////////////////////////////////
// Setup Vertex Data, Buffers and configure Vertex Attributes:

  float vertices[] = {
   // pos (x,y,z)      // text-coords (u,v)
   -1.0f, -1.0f, 0.0f, 0.0f, 0.0f, // SW
    1.0f, -1.0f, 0.0f, 1.0f, 0.0f, // SE
   -1.0f,  1.0f, 0.0f, 0.0f, 1.0f, // NW
    1.0f,  1.0f, 0.0f, 1.0f, 1.0f  // NE
  };

  // generate buffer and Array for vertices and bind and fill it
  GLuint VBO;// Vertex Buffer Object, Vertex Array Object
  glGenBuffers(1, &VBO);
  glBindBuffer(GL_ARRAY_BUFFER, VBO);
  // Copy Vertices Data
  glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);
  // Explain Data via VertexAttributePointers to the shader
  // By Default, it's disabled
  // Enable the Vertex Attribute
  glEnableVertexAttribArray(GL_VERTEX_POSITION_ATTRIBUTE);
  glVertexAttribPointer(GL_VERTEX_POSITION_ATTRIBUTE, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (GLvoid*)0);
  // Same for Texture: Pay Attention of Stride and begin
  glEnableVertexAttribArray(GL_VERTEX_TEXTURE_COORD_ATTRIBUTE);
  glVertexAttribPointer(GL_VERTEX_TEXTURE_COORD_ATTRIBUTE, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (GLvoid*)(3 * sizeof(float)));

  // possible unbinding:
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindVertexArray(0);

  // uncomment this call to draw in wireframe polygons.
  //glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

///////////////////////////////////////////////////////////////////////////////
// CUDA Texture Interaction

  GLuint interopTexture;
  unsigned char *deviceTextureGraphic;
  size_t deviceTextureGraphicPitch;
  hipGraphicsResource *textureGraphicResource;
  hipArray *textureCudaArray;


  // calculate Data size and MemAlloc Cuda Buffer
  size_t textureWidth = 2;
  size_t textureHeight = 2;
  //int numTexels = textureHeight * textureWidth;
  //int numValues = numTexels*4; // RGBA
  //size_t sizeTexData = numValues * sizeof(GLubyte);
  checkCuda( hipMallocPitch(&deviceTextureGraphic, &deviceTextureGraphicPitch, textureWidth * 4, textureHeight) );
  printf("[LOG] Allocated Texture Memory: %zu x %zu with pitch: %zu\n", textureWidth, textureHeight, deviceTextureGraphicPitch);
  printf("[LOG] size of pixelRGBA: %zu\n", sizeof(pixelRGBA));
  
  // Here the Calculations for the interop-Data
  glGenTextures(1, &interopTexture);
  glBindTexture(GL_TEXTURE_2D, interopTexture);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);	
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  // Just allocate, but no copy to it:
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, textureWidth, textureHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
  // Register OpenGL texture to a hipGraphicsResource for CUDA
  checkCuda( hipGraphicsGLRegisterImage( &textureGraphicResource, interopTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

///////////////////////////////////////////////////////////////////////////////
// Render Loop
  double lastTime = glfwGetTime();
  int nbFrames = 0;
  char windowTitle[100];

  while (!glfwWindowShouldClose(window))
  {
    // Measure speed
    double currentTime = glfwGetTime();
    nbFrames++;
    if ( currentTime - lastTime >= 1.0 ){ // If last prinf() was more than 1 sec ago
        // printf and reset timer
        sprintf(windowTitle, "Hello Cuda GLFW Interop | %f ms/frame", 1000.0/double(nbFrames));
        glfwSetWindowTitle(window, windowTitle);
        nbFrames = 0;
        lastTime += 1.0;
    }

    // set bg color here via Clearing
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    // run CUDA
    myTextureKernel<<<1, 1>>>(deviceTextureGraphic, textureWidth, textureHeight, deviceTextureGraphicPitch);
    // Map 1 graphics resource for access by CUDA in stream 0
    checkCuda( hipGraphicsMapResources(1, &textureGraphicResource, 0) );
    // get the corresponding CudaArray of Resource at array position 0 and mipmap level 0
    checkCuda( hipGraphicsSubResourceGetMappedArray(&textureCudaArray, textureGraphicResource, 0, 0) );
    // copy Data to CudaArray from deviceRenderBuffer, wOffset=0, hOffset=0
    //checkCuda( hipMemcpyToArray(textureCudaArray, 0, 0, deviceRenderBuffer, sizeTexData, hipMemcpyDeviceToDevice) ); // deprecated
    checkCuda( hipMemcpy2DToArray(textureCudaArray, 0, 0, deviceTextureGraphic, deviceTextureGraphicPitch, textureWidth*4, textureHeight, hipMemcpyDeviceToDevice));
    // Unmap 1 resource from Stream 0
    checkCuda( hipGraphicsUnmapResources(1, &textureGraphicResource, 0) );

    // Draw
    // Use the program for the pipeline (keep it to save state to VAO)
    glUseProgram(shaderProgram);
    glBindVertexArray(VAO); // Program is bound to VAO
    glDrawArrays(GL_TRIANGLE_STRIP, 0, 4); // All about the loaded VAO
    glBindVertexArray(0); // To unbind Vertex Array
    glUseProgram(0);

    // Swap front and back buffers
    glfwSwapBuffers(window);

    // Check for Inputs:
    processInput(window);
    // Poll for and process events
    glfwPollEvents();
  }

  // Cleanup
  // OpenGL is reference counted and terminated by GLFW
  checkCuda( hipGraphicsUnregisterResource(textureGraphicResource) );
  checkCuda( hipFree(deviceTextureGraphic) );
  glfwDestroyWindow(window);
  glfwTerminate();
  return EXIT_SUCCESS;
}
